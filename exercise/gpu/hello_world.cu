#include <hip/hip_runtime.h>
#include <stdio.h>

#define K 2






__global__ void hello() {
	auto tid = threadIdx.x;
	auto bid = blockIdx.x;
	printf("tid=%u running from bid=%u\n", tid, bid);
}

int main() {
	hello<<< 1, 512 >>>();

	hipDeviceSynchronize();


}
